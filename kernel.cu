#include "hip/hip_runtime.h"
﻿#include "Cuda.h"
#include "hip/hip_runtime.h"
#include ""

#if _DEBUG
#   define cudaCall(cuda_func, ...) { hipError_t status = cuda_func(__VA_ARGS__); cudaAssert((status), __FILE__, #cuda_func, __LINE__); }
#else
#   define cudaCall(cuda_func, ...) { hipError_t status = cuda_func(__VA_ARGS__); }
#endif

inline void cudaAssert(hipError_t status, const char* file, const char* func, int line)
{
    if (status != hipSuccess)
    {
        std::stringstream ss;
        ss << "Error: " << hipGetErrorString(status) << std::endl;
        ss << "Func: " << func << std::endl;
        ss << "File: " << file << std::endl;
        ss << "Line: " << line << std::endl;

        throw std::runtime_error(ss.str());
    }
}

__device__ double* escape_count_ssaa;
__device__ double* magnitude_ssaa;
__device__ double* escape_count;

double* escape_count_ssaa_d;
double* magnitude_ssaa_d;
double* escape_count_d;

__global__ void downsample(
    double* escape_count_ssaa,
    double* magnitude_ssaa,
    double* escape_count,
    const std::uint32_t max_iterations,
    const int width,
    const int height,
    const int ssaa,
    const int smooth_shading)
{
    const int y_pixel = threadIdx.y + blockIdx.y * blockDim.y;
    const int x_pixel = threadIdx.x + blockIdx.x * blockDim.x;

    int acccessor = y_pixel * width + x_pixel;

    // Pixel
    int index;
    int smallestIndex;
    double min;
    double norm;

    min = max_iterations;
    smallestIndex = ((y_pixel * ssaa + 0) * width * ssaa) + (x_pixel * ssaa + 0);
    // Sub Pixel
    for (int y = 0; y < ssaa; ++y)
    {
        for (int x = 0; x < ssaa; ++x)
        {
            index = ((y_pixel * ssaa + y) * width * ssaa) + (x_pixel * ssaa + x);
            if (escape_count_ssaa[index] < min)
            {
                smallestIndex = index;
                min = escape_count_ssaa[index];
            }
        }
    }

    if (smooth_shading == 1)
    {
        norm = min + 1 - log2f(log2f(sqrt(magnitude_ssaa[smallestIndex]))) / log2f(2);
        if (norm > max_iterations || norm < -1000 || min == max_iterations - 1)
        {
            norm = max_iterations;
        }

        escape_count[acccessor] = norm;
    }
    else
    {
        escape_count[acccessor] = min;
    }
}

__global__ void mandelbrot_kernel(
    double* escape_count,
    double* magnitude,
    const std::uint32_t max_iterations,
    const int width, 
    const int height, 
    const double x_scale,
    const double y_scale,
    const double x_min, 
    const double y_min)
{
    const int y_pixel = threadIdx.y + blockIdx.y * blockDim.y;
    const int x_pixel = threadIdx.x + blockIdx.x * blockDim.x;

    int acccessor = y_pixel * width + x_pixel;

    if (y_pixel >= height || x_pixel >= width)
    {
        return;
    }

    double y = y_min + (y_pixel * x_scale);
    double x = x_min + (x_pixel * y_scale);

    double yy = y * y;
    double zx = (x - 0.25) * (x - 0.25) + yy;

    if (zx*(zx + x - 0.25) - 0.25 * yy < 0 || (x + 1.0) * (x + 1.0) + yy - 0.0625 < 0)
    {
        escape_count[acccessor] = max_iterations;
        magnitude[acccessor] = INFINITY;
        return;
    }

    std::uint32_t curr_iterations = 0;
    double zy, zx2, zy2, mag;
    zx = zy = zx2 = zy2 = mag = 0.0;

    do {
        zy = 2.0 * zx * zy + y;
        zx = zx2 - zy2 + x;
        zx2 = zx * zx;
        zy2 = zy * zy;
        mag = zx2 + zy2;
    } while (curr_iterations++ < max_iterations && mag < 4.0);

    escape_count[acccessor] = curr_iterations;
    magnitude[acccessor] = mag;
}

namespace mandelbrot_cuda
{
    bool InitCUDA(Properties* properties)
    {
        const int ssaa = properties->Get_Mandelbrot_SuperSamplingAntiAliasing();
        const int width = properties->Get_General_Width();
        const int height = properties->Get_General_Height();

        const int width_ssaa = width * ssaa;
        const int height_ssaa = height * ssaa;

        int count = 0;
        int i = 0;

        cudaCall(hipGetDeviceCount, &count);
        if (count == 0) {
            fprintf(stderr, "There is no device.\n");
            return false;
        }

        for (i = 0; i < count; i++) {
            hipDeviceProp_t prop;
            if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
                if (prop.major >= 1) {
                    break;
                }
            }
        }
        if (i == count) {
            fprintf(stderr, "There is no device supporting CUDA.\n");
            return false;
        }
        cudaCall(hipSetDevice, i);
        cudaCall(hipFree, 0);

        const size_t buffer_size_ssaa = width_ssaa * height_ssaa * sizeof(double);
        const size_t buffer_size = width * height * sizeof(double);

        // Allocate memory
        hipMalloc((void**)&escape_count_ssaa_d, buffer_size_ssaa);
        hipMalloc((void**)&magnitude_ssaa_d, buffer_size_ssaa);
        hipHostMalloc((void**)&escape_count_d, buffer_size);

        // Zero memory and assign to global device symbol
        //hipMemset(escape_count_ssaa_d, 0, buffer_size_ssaa);
        //hipMemset(magnitude_ssaa_d, 0, buffer_size_ssaa);
        //hipMemset(escape_count_d, 0, buffer_size);
        cudaCall(hipMemcpyToSymbol, escape_count_ssaa, &escape_count_ssaa_d, sizeof(double*));
        cudaCall(hipMemcpyToSymbol, magnitude_ssaa, &magnitude_ssaa_d, sizeof(double*));
        cudaCall(hipMemcpyToSymbol, escape_count, &escape_count_d, sizeof(double*));

        printf("CUDA initialized.\n");
        return true;
    }

    bool DestroyCUDA() {
        cudaCall(hipFree, escape_count_ssaa_d);
        cudaCall(hipFree, magnitude_ssaa_d);
        cudaCall(hipHostFree, escape_count_d);

        printf("CUDA destroyed.\n");
        return true;
    }

    template<class T, typename... A>
    float launch_kernel(T& kernel, dim3 work, A&&... args)
    {
        int device;
        hipDeviceProp_t props;
        hipGetDevice(&device);
        hipGetDeviceProperties(&props, device);

        int threadBlocks;
        if (props.major == 2)
        {
            threadBlocks = 8;
        }
        else if (props.major == 3)
        {
            threadBlocks = 16;
        }
        else
        {
            threadBlocks = 32;
        }

        int blockSize;
        std::uint32_t minGridSize;
        hipOccupancyMaxPotentialBlockSize((int*)&minGridSize, &blockSize, kernel, 0, 0);

        int maxActiveBlocks = 0;
        do
        {
            hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, kernel, blockSize, 0);

            if (blockSize < props.warpSize || maxActiveBlocks >= threadBlocks)
            {
                break;
            }

            blockSize -= props.warpSize;
        } while (true);

        int blockSizeDimX, blockSizeDimY;
        blockSizeDimX = blockSizeDimY = (int)pow(2, ceil(log(sqrt(blockSize)) / log(2)));

        while (blockSizeDimX * blockSizeDimY > blockSize)
        {
            blockSizeDimY--;
        }

        dim3 block(blockSizeDimX, blockSizeDimY);
        dim3 grid((work.x + block.x - 1) / block.x, (work.y + block.y - 1) / block.y);
        grid.x = grid.x > minGridSize ? grid.x : minGridSize;
        grid.y = grid.y > minGridSize ? grid.y : minGridSize;


        float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / (float)(props.maxThreadsPerMultiProcessor / props.warpSize);

        std::cout << "Grid of size " << grid.x * grid.y << std::endl;
        std::cout << "Launched blocks of size " << blockSize << std::endl;
        std::cout << "Theoretical occupancy " << occupancy * 100.0f << "%" << std::endl;


        hipEvent_t start;
        cudaCall(hipEventCreate, &start);

        hipEvent_t stop;
        cudaCall(hipEventCreate, &stop);

        cudaCall(hipEventRecord, start, 0);

        kernel << < grid, block >> > (std::forward<A>(args)...);

        cudaCall(hipGetLastError);
        cudaCall(hipEventRecord, stop, 0);
        cudaCall(hipEventSynchronize, stop);

        float elapsed_time;
        cudaCall(hipEventElapsedTime, &elapsed_time, start, stop);

        cudaCall(hipEventDestroy, start);
        cudaCall(hipEventDestroy, stop);

        hipProfilerStop();

        return elapsed_time;
    }

    std::uint64_t generate_mandelbrot(double* escape_count_ret, Properties* properties)
    {
        const int width = properties->Get_General_Width();
        const int height = properties->Get_General_Height();
        const int ssaa = properties->Get_Mandelbrot_SuperSamplingAntiAliasing();
        const int max_iterations = properties->Get_Mandelbrot_MaxIterations();
        double x_min = properties->Get_Mandelbrot_Point_Cr() - properties->Get_Mandelbrot_Point_Rr();
        double x_max = properties->Get_Mandelbrot_Point_Cr() + properties->Get_Mandelbrot_Point_Rr();
        double y_min = properties->Get_Mandelbrot_Point_Ci() - properties->Get_Mandelbrot_Point_Rr() * ((double)height / width);
        double y_max = properties->Get_Mandelbrot_Point_Ci() + properties->Get_Mandelbrot_Point_Rr() * ((double)height / width);
        const int smooth_shading = properties->Get_Mandelbrot_SmoothShading();

        const int width_ssaa = width * ssaa;
        const int height_ssaa = height * ssaa;

        const double x_scale = (x_max - x_min) / width_ssaa;
        const double y_scale = (y_max - y_min) / height_ssaa;

        const int size_d = width * height * sizeof(double);

        float elapsed_time = launch_kernel(mandelbrot_kernel, 
            dim3(width_ssaa, height_ssaa),
            escape_count_ssaa_d,
            magnitude_ssaa_d,
            max_iterations, 
            width_ssaa,
            height_ssaa,
            x_scale,
            y_scale,
            x_min,
            y_min);

        elapsed_time += launch_kernel(downsample,
            dim3(width, height),
            escape_count_ssaa_d,
            magnitude_ssaa_d,
            escape_count_d,
            max_iterations,
            width,
            height,
            ssaa,
            smooth_shading);

        cudaCall(hipMemcpy, escape_count_ret, escape_count_d, size_d, hipMemcpyDeviceToHost);

        return static_cast<std::uint64_t>(elapsed_time);
    }
}